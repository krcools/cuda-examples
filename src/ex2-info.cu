#include "hip/hip_runtime.h"
#include <stdio.h>
 

__global__ void add(int a, int b, int *c) 
{
	//a[threadIdx.x] += b[threadIdx.x];
	*c = a + b;
}
 
int main()
{
	int c;
	int *dev_c;
	hipMalloc( (void**)&dev_c, sizeof(int) );
	add << <1, 1 >> > (2, 7, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("2+7 = %d\n", c);
	hipFree(dev_c);

	int count;
	hipGetDeviceCount(&count);
	printf("Number of devices: %d\n", count);

	hipDeviceProp_t prop;
	for (int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);
		printf("Device %d:\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);
		printf("Global memory: %ld\n", prop.totalGlobalMem);
		printf("MP count: %d\n", prop.multiProcessorCount);
		printf("Threads in warp: %d", prop.warpSize);
	}
}